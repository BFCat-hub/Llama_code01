#include <hip/hip_runtime.h>

#include <stdio.h>
#include <cmath>

__global__ void pow_kernel(int N, float ALPHA, float* X, int INCX, float* Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) {
        Y[i * INCY] = pow(X[i * INCX], ALPHA);
    }
}

int main() {
    // 定义数组大小
    const int N = 1000;

    // 分配主机端内存
    float* h_X = (float*)malloc(N * sizeof(float));
    float* h_Y = (float*)malloc(N * sizeof(float));

    // 初始化数组数据
    for (int i = 0; i < N; ++i) {
        h_X[i] = static_cast<float>(i);
    }

    // 分配设备端内存
    float* d_X;
    float* d_Y;
    hipMalloc((void**)&d_X, N * sizeof(float));
    hipMalloc((void**)&d_Y, N * sizeof(float));

    // 将数组数据从主机端拷贝到设备端
    hipMemcpy(d_X, h_X, N * sizeof(float), hipMemcpyHostToDevice);

    // 设置线程块和网格大小
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, 1);

    // 启动内核
    pow_kernel<<<gridSize, blockSize>>>(N, 2.0, d_X, 1, d_Y, 1);

    // 将结果从设备端拷贝回主机端
    hipMemcpy(h_Y, d_Y, N * sizeof(float), hipMemcpyDeviceToHost);

    // 打印部分结果
    for (int i = 0; i < 10; ++i) {
        printf("Y[%d]: %f\n", i, h_Y[i]);
    }

    // 释放内存
    free(h_X);
    free(h_Y);
    hipFree(d_X);
    hipFree(d_Y);

    return 0;
}
