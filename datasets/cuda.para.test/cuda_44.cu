#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void gpu_add(float* c, float* a, float* b, int n) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int m = gridDim.x * blockDim.x;
    for (int k = j; k < n; k += m) {
        c[k] = a[k] + b[k];
    }
}

int main() {
    // 定义数组大小
    const int arraySize = 1000;

    // 分配主机端内存
    float* h_a = (float*)malloc(arraySize * sizeof(float));
    float* h_b = (float*)malloc(arraySize * sizeof(float));
    float* h_c = (float*)malloc(arraySize * sizeof(float));

    // 初始化数组数据
    for (int i = 0; i < arraySize; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(2 * i);
    }

    // 分配设备端内存
    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc((void**)&d_a, arraySize * sizeof(float));
    hipMalloc((void**)&d_b, arraySize * sizeof(float));
    hipMalloc((void**)&d_c, arraySize * sizeof(float));

    // 将数组数据从主机端拷贝到设备端
    hipMemcpy(d_a, h_a, arraySize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, arraySize * sizeof(float), hipMemcpyHostToDevice);

    // 设置线程块和网格大小
    dim3 blockSize(256);
    dim3 gridSize((arraySize + blockSize.x - 1) / blockSize.x, 1);

    // 启动内核
    gpu_add<<<gridSize, blockSize>>>(d_c, d_a, d_b, arraySize);

    // 将结果从设备端拷贝回主机端
    hipMemcpy(h_c, d_c, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    // 打印部分结果
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_c[i]);
    }

    // 释放内存
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
