#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void dotKernel(float* c, float* a, float* b) {
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    c[t_id] = a[t_id] * b[t_id];
}

int main() {
    // 设置数组大小
    int arraySize = 1000;

    // 分配主机端内存
    float* h_a = (float*)malloc(arraySize * sizeof(float));
    float* h_b = (float*)malloc(arraySize * sizeof(float));
    float* h_c = (float*)malloc(arraySize * sizeof(float));

    // 初始化数据
    for (int i = 0; i < arraySize; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(2 * i);
    }

    // 分配设备端内存
    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc((void**)&d_a, arraySize * sizeof(float));
    hipMalloc((void**)&d_b, arraySize * sizeof(float));
    hipMalloc((void**)&d_c, arraySize * sizeof(float));

    // 将数据从主机端拷贝到设备端
    hipMemcpy(d_a, h_a, arraySize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, arraySize * sizeof(float), hipMemcpyHostToDevice);

    // 设置线程块和网格大小
    int blockSize = 256;
    int gridSize = (arraySize + blockSize - 1) / blockSize;

    // 启动内核
    dotKernel<<<gridSize, blockSize>>>(d_c, d_a, d_b);

    // 将结果从设备端拷贝回主机端
    hipMemcpy(h_c, d_c, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果（为了简化，此处不打印全部结果）
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_c[i]);
    }

    // 释放内存
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
