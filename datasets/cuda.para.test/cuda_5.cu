#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void scale_dev(float* array, float scale, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        array[idx] *= scale;
    }
}

int main() {
    // 设置数组大小
    int arraySize = 1000;

    // 设置缩放因子
    float scale = 1.5f;

    // 分配主机端内存
    float* h_array = (float*)malloc(arraySize * sizeof(float));

    // 初始化数据
    for (int i = 0; i < arraySize; ++i) {
        h_array[i] = static_cast<float>(i);
    }

    // 分配设备端内存
    float* d_array;
    hipMalloc((void**)&d_array, arraySize * sizeof(float));

    // 将数据从主机端拷贝到设备端
    hipMemcpy(d_array, h_array, arraySize * sizeof(float), hipMemcpyHostToDevice);

    // 设置线程块和网格大小
    int blockSize = 256;
    int gridSize = (arraySize + blockSize - 1) / blockSize;

    // 启动内核
    scale_dev<<<gridSize, blockSize>>>(d_array, scale, arraySize);

    // 将结果从设备端拷贝回主机端
    hipMemcpy(h_array, d_array, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果（为了简化，此处不打印全部结果）
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_array[i]);
    }

    // 释放内存
    free(h_array);
    hipFree(d_array);

    return 0;
}
