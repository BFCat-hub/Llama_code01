
#include <hip/hip_runtime.h>

#include <stdio.h>

// Define the CUDA kernel
__global__ void ConvLayerForward_Kernel(int C, int W_grid, int K, float *X, float *W, float *Y) {
    int n, m, h, w, c, p, q;
    
    n = blockIdx.x;
    m = blockIdx.y;
    h = blockIdx.z / W_grid + threadIdx.y;
    w = blockIdx.z % W_grid + threadIdx.x;

    float acc = 0;

    for (c = 0; c < C; c++) {
        for (p = 0; p < K; p++) {
            for (q = 0; q < K; q++) {
                acc += X[n * C * W_grid * W_grid + c * W_grid * W_grid + (h + p) * W_grid + (w + q)] * W[m * C * K * K + c * K * K + p * K + q];
            }
        }
    }

    Y[n * W_grid * W_grid * W_grid + m * W_grid * W_grid + h * W_grid + w] = acc;
}

int main() {
    // Example usage
    int C = 3, W_grid = 4, K = 3;  // Set your values accordingly
    float *X, *W, *Y;  // Assuming these arrays are allocated and initialized

    // Set the CUDA device
    hipSetDevice(0);

    // Allocate device memory
    float *d_X, *d_W, *d_Y;
    hipMalloc((void **)&d_X, C * W_grid * W_grid * sizeof(float));
    hipMalloc((void **)&d_W, C * K * K * sizeof(float));
    hipMalloc((void **)&d_Y, W_grid * W_grid * W_grid * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_X, X, C * W_grid * W_grid * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W, W, C * K * K * sizeof(float), hipMemcpyHostToDevice);

    // Configure the CUDA kernel launch
    dim3 threadsPerBlock(K, K);
    dim3 blocksPerGrid(W_grid, W_grid, W_grid);

    // Launch the CUDA kernel
    ConvLayerForward_Kernel<<<blocksPerGrid, threadsPerBlock>>>(C, W_grid, K, d_X, d_W, d_Y);

    // Wait for the kernel to finish
    hipDeviceSynchronize();

    // Copy the result back to the host
    hipMemcpy(Y, d_Y, W_grid * W_grid * W_grid * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_X);
    hipFree(d_W);
    hipFree(d_Y);

    return 0;
}
