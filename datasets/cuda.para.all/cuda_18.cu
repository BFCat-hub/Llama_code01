#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void gpu_add(float* c, float* a, float* b, int n) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < n) {
        c[j] = a[j] + b[j];
    }
}

int main() {
    // 设置数组大小
    int arraySize = 1000;

    // 分配主机端内存
    float* h_a = (float*)malloc(arraySize * sizeof(float));
    float* h_b = (float*)malloc(arraySize * sizeof(float));
    float* h_c = (float*)malloc(arraySize * sizeof(float));

    // 初始化数据
    for (int i = 0; i < arraySize; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(2 * i);
    }

    // 分配设备端内存
    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc((void**)&d_a, arraySize * sizeof(float));
    hipMalloc((void**)&d_b, arraySize * sizeof(float));
    hipMalloc((void**)&d_c, arraySize * sizeof(float));

    // 将数据从主机端拷贝到设备端
    hipMemcpy(d_a, h_a, arraySize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, arraySize * sizeof(float), hipMemcpyHostToDevice);

    // 设置线程块和网格大小
    int blockSize = 256;
    int gridSize = (arraySize + blockSize - 1) / blockSize;

    // 启动内核
    gpu_add<<<gridSize, blockSize>>>(d_c, d_a, d_b, arraySize);

    // 将结果从设备端拷贝回主机端
    hipMemcpy(h_c, d_c, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果（为了简化，此处不打印全部结果）
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_c[i]);
    }

    // 释放内存
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
