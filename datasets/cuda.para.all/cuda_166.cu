
#include <hip/hip_runtime.h>

#include <stdio.h>

// Define the CUDA kernel
__global__ void get_before_nms_data(const float *boxes, const float *scores, const int *labels, const int *index,
                                     float *boxes_out, float *scores_out, int *labels_out, int dims) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= dims) {
        return;
    }

    if (index[tid] == 0) {
        boxes_out[tid * 4 + 0] = -1;
        boxes_out[tid * 4 + 1] = -1;
        boxes_out[tid * 4 + 2] = -1;
        boxes_out[tid * 4 + 3] = -1;
        scores_out[tid] = -1;
        labels_out[tid] = -1;
    } else {
        boxes_out[tid * 4 + 0] = boxes[tid * 4 + 0];
        boxes_out[tid * 4 + 1] = boxes[tid * 4 + 1];
        boxes_out[tid * 4 + 2] = boxes[tid * 4 + 2];
        boxes_out[tid * 4 + 3] = boxes[tid * 4 + 3];
        scores_out[tid] = scores[tid];
        labels_out[tid] = labels[tid];
    }
}

int main() {
    // Example usage
    int dims = 1000;  // Set your value of dims accordingly
    float *boxes, *scores, *boxes_out, *scores_out;  // Assuming these arrays are allocated and initialized
    int *labels, *index, *labels_out;                // Assuming these arrays are allocated and initialized

    // Set the CUDA device
    hipSetDevice(0);

    // Allocate device memory
    float *d_boxes, *d_scores, *d_boxes_out, *d_scores_out;
    int *d_labels, *d_index, *d_labels_out;

    hipMalloc((void **)&d_boxes, dims * 4 * sizeof(float));
    hipMalloc((void **)&d_scores, dims * sizeof(float));
    hipMalloc((void **)&d_labels, dims * sizeof(int));
    hipMalloc((void **)&d_index, dims * sizeof(int));
    hipMalloc((void **)&d_boxes_out, dims * 4 * sizeof(float));
    hipMalloc((void **)&d_scores_out, dims * sizeof(float));
    hipMalloc((void **)&d_labels_out, dims * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_boxes, boxes, dims * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_scores, scores, dims * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_labels, labels, dims * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_index, index, dims * sizeof(int), hipMemcpyHostToDevice);

    // Configure the CUDA kernel launch
    int threadsPerBlock = 256;
    int blocksPerGrid = (dims + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the CUDA kernel
    get_before_nms_data<<<blocksPerGrid, threadsPerBlock>>>(d_boxes, d_scores, d_labels, d_index,
                                                            d_boxes_out, d_scores_out, d_labels_out, dims);

    // Wait for the kernel to finish
    hipDeviceSynchronize();

    // Copy the result back to the host
    hipMemcpy(boxes_out, d_boxes_out, dims * 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(scores_out, d_scores_out, dims * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(labels_out, d_labels_out, dims * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_boxes);
    hipFree(d_scores);
    hipFree(d_labels);
    hipFree(d_index);
    hipFree(d_boxes_out);
    hipFree(d_scores_out);
    hipFree(d_labels_out);

    return 0;
}
