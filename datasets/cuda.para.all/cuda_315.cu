
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512
#define Size 1024

// CUDA kernel
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary) {
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    if (s >= size)
        return;

    int i = 0;
    float mean = 0;

    for (i = 0; i < n; ++i) {
        mean += fabs(input[i * size + s]);
    }

    mean = mean / n;

    for (i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}

int main() {
    // Allocate device memory
    float *d_input, *d_binary;

    hipMalloc((void **)&d_input, N * Size * sizeof(float));
    hipMalloc((void **)&d_binary, N * Size * sizeof(float));

    // Set grid and block sizes
    dim3 blockSize(256);  // You may adjust the block size
    dim3 gridSize((N * Size + blockSize.x - 1) / blockSize.x);

    // Launch the kernel
    binarize_input_kernel<<<gridSize, blockSize>>>(d_input, N, Size, d_binary);

    // Synchronize to make sure the kernel finishes before proceeding
    hipDeviceSynchronize();

    // Check for errors during the kernel launch
    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaErr));
        return 1;
    }

    // Cleanup
    hipFree(d_input);
    hipFree(d_binary);

    return 0;
}
 
