
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA核函数
__global__ void vectorDiv(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] / B[i];
    }
}

int main() {
    // 设置数组大小
    const int array_size = 100;

    // 在设备上分配空间
    float *A_device, *B_device, *C_device;

    hipMalloc((void**)&A_device, array_size * sizeof(float));
    hipMalloc((void**)&B_device, array_size * sizeof(float));
    hipMalloc((void**)&C_device, array_size * sizeof(float));

    // 初始化输入数组数据
    float *A_host = (float *)malloc(array_size * sizeof(float));
    float *B_host = (float *)malloc(array_size * sizeof(float));
    float *C_host = (float *)malloc(array_size * sizeof(float));

    for (int i = 0; i < array_size; ++i) {
        A_host[i] = i + 1.0; // 为了演示目的，将输入数据初始化为 1.0, 2.0, 3.0, ...
        B_host[i] = (i + 1.0) * 2.0; // 为了演示目的，将输入数据初始化为 2.0, 4.0, 6.0, ...
        C_host[i] = 0.0; // 初始化为0.0
    }

    // 将输入数据从主机复制到设备
    hipMemcpy(A_device, A_host, array_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_device, B_host, array_size * sizeof(float), hipMemcpyHostToDevice);

    // 定义启动配置
    dim3 blockDim(256); // 块大小为256个线程
    dim3 gridDim((array_size + blockDim.x - 1) / blockDim.x); // 确保足够的块数

    // 调用CUDA核函数
    vectorDiv<<<gridDim, blockDim>>>(A_device, B_device, C_device, array_size);

    // 将结果从设备复制回主机
    hipMemcpy(C_host, C_device, array_size * sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果
    printf("Result after CUDA kernel execution:\n");
    for (int i = 0; i < array_size; ++i) {
        printf("%.2f ", C_host[i]);
    }
    printf("\n");

    // 释放内存
    free(A_host);
    free(B_host);
    free(C_host);
    hipFree(A_device);
    hipFree(B_device);
    hipFree(C_device);

    return 0;
}
 
