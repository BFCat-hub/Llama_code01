
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel function
__global__ void doubleArrayVectorElementwiseMultiplyKernel(double* d_in_a, double* d_in_b, double* d_out, int length) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < length) {
        d_out[tid] = d_in_a[tid] * d_in_b[tid];
    }
}

int main() {
    // Vector size
    int length = 10; // Change this according to your requirements

    // Host arrays
    double* h_in_a = (double*)malloc(length * sizeof(double));
    double* h_in_b = (double*)malloc(length * sizeof(double));
    double* h_out = (double*)malloc(length * sizeof(double));

    // Initialize host input arrays
    for (int i = 0; i < length; ++i) {
        h_in_a[i] = static_cast<double>(i + 1); // Example data, you can modify this accordingly
        h_in_b[i] = static_cast<double>(i);
    }

    // Device arrays
    double* d_in_a;
    double* d_in_b;
    double* d_out;
    hipMalloc((void**)&d_in_a, length * sizeof(double));
    hipMalloc((void**)&d_in_b, length * sizeof(double));
    hipMalloc((void**)&d_out, length * sizeof(double));

    // Copy host input arrays to device
    hipMemcpy(d_in_a, h_in_a, length * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_in_b, h_in_b, length * sizeof(double), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    int block_size = 256;
    dim3 grid_size((length + block_size - 1) / block_size, 1);

    // Launch the CUDA kernel
    doubleArrayVectorElementwiseMultiplyKernel<<<grid_size, block_size>>>(d_in_a, d_in_b, d_out, length);

    // Copy the result back to the host
    hipMemcpy(h_out, d_out, length * sizeof(double), hipMemcpyDeviceToHost);

    // Display the result
    printf("Result: ");
    for (int i = 0; i < length; ++i) {
        printf("%f ", h_out[i]);
    }
    printf("\n");

    // Clean up
    free(h_in_a);
    free(h_in_b);
    free(h_out);
    hipFree(d_in_a);
    hipFree(d_in_b);
    hipFree(d_out);

    return 0;
}
 
