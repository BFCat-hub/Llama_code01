#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void dmul_Scalar_matrix(double* a, double value, double* c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] * value;
    }
}

int main() {
    // 设置数组大小
    int arraySize = 1000;

    // 设置缩放因子
    double value = 2.0;

    // 分配主机端内存
    double* h_a = (double*)malloc(arraySize * sizeof(double));
    double* h_c = (double*)malloc(arraySize * sizeof(double));

    // 初始化数据
    for (int i = 0; i < arraySize; ++i) {
        h_a[i] = static_cast<double>(i);
    }

    // 分配设备端内存
    double* d_a;
    double* d_c;
    hipMalloc((void**)&d_a, arraySize * sizeof(double));
    hipMalloc((void**)&d_c, arraySize * sizeof(double));

    // 将数据从主机端拷贝到设备端
    hipMemcpy(d_a, h_a, arraySize * sizeof(double), hipMemcpyHostToDevice);

    // 设置线程块和网格大小
    int blockSize = 256;
    int gridSize = (arraySize + blockSize - 1) / blockSize;

    // 启动内核
    dmul_Scalar_matrix<<<gridSize, blockSize>>>(d_a, value, d_c, arraySize);

    // 将结果从设备端拷贝回主机端
    hipMemcpy(h_c, d_c, arraySize * sizeof(double), hipMemcpyDeviceToHost);

    // 打印结果（为了简化，此处不打印全部结果）
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_c[i]);
    }

    // 释放内存
    free(h_a);
    free(h_c);
    hipFree(d_a);
    hipFree(d_c);

    return 0;
}
