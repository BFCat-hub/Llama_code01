
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel function
__global__ void gpuSearchPosShmem1EQ(int key, int* devKey, int* devPos, int size) {
    int globalTx = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalTx < size) {
        if (devKey[globalTx] == key) {
            devPos[0] = globalTx;
        }
    }
}

int main() {
    // Array size and search key
    int size = 10; // Change this according to your requirements
    int key = 5;   // Change this according to your requirements

    // Host arrays
    int* h_devKey = (int*)malloc(size * sizeof(int));
    int* h_devPos = (int*)malloc(sizeof(int));

    // Initialize host input array
    for (int i = 0; i < size; ++i) {
        h_devKey[i] = i; // Example data, you can modify this accordingly
    }

    // Device arrays
    int* d_devKey;
    int* d_devPos;
    hipMalloc((void**)&d_devKey, size * sizeof(int));
    hipMalloc((void**)&d_devPos, sizeof(int));

    // Copy host input array to device
    hipMemcpy(d_devKey, h_devKey, size * sizeof(int), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    int block_size = 256;
    dim3 grid_size((size + block_size - 1) / block_size, 1);

    // Launch the CUDA kernel
    gpuSearchPosShmem1EQ<<<grid_size, block_size>>>(key, d_devKey, d_devPos, size);

    // Copy the result back to the host
    hipMemcpy(h_devPos, d_devPos, sizeof(int), hipMemcpyDeviceToHost);

    // Display the result
    printf("Position of key %d: %d\n", key, h_devPos[0]);

    // Clean up
    free(h_devKey);
    free(h_devPos);
    hipFree(d_devKey);
    hipFree(d_devPos);

    return 0;
}
 
