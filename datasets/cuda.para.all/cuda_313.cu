
 #include <stdio.h>
#include <hip/hip_runtime.h>

#define DIM 128
#define N 1024

// CUDA kernel
__global__ void solveLowerKernel(const double *lower, const double *b, double *buf, int dim, int n) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k < n) {
        for (int i = 0; i < dim; i++) {
            double val = b[k * dim + i];
            for (int j = 0; j < i; j++) {
                val -= lower[i * dim + j] * buf[k * dim + j];
            }
            buf[k * dim + i] = val / lower[i * dim + i];
        }
    }
}

int main() {
    // Allocate device memory
    double *d_lower, *d_b, *d_buf;

    hipMalloc((void **)&d_lower, DIM * DIM * sizeof(double));
    hipMalloc((void **)&d_b, N * DIM * sizeof(double));
    hipMalloc((void **)&d_buf, N * DIM * sizeof(double));

    // Set grid and block sizes
    dim3 blockSize(256);  // You may adjust the block size
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);

    // Launch the kernel
    solveLowerKernel<<<gridSize, blockSize>>>(d_lower, d_b, d_buf, DIM, N);

    // Synchronize to make sure the kernel finishes before proceeding
    hipDeviceSynchronize();

    // Check for errors during the kernel launch
    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaErr));
        return 1;
    }

    // Cleanup
    hipFree(d_lower);
    hipFree(d_b);
    hipFree(d_buf);

    return 0;
}

