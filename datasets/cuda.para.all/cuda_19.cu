#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void mul_Scalar_matrix(float* a, float value, float* c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] * value;
    }
}

int main() {
    // 设置数组大小
    int arraySize = 1000;

    // 设置缩放因子
    float value = 2.0;

    // 分配主机端内存
    float* h_a = (float*)malloc(arraySize * sizeof(float));
    float* h_c = (float*)malloc(arraySize * sizeof(float));

    // 初始化数据
    for (int i = 0; i < arraySize; ++i) {
        h_a[i] = static_cast<float>(i);
    }

    // 分配设备端内存
    float* d_a;
    float* d_c;
    hipMalloc((void**)&d_a, arraySize * sizeof(float));
    hipMalloc((void**)&d_c, arraySize * sizeof(float));

    // 将数据从主机端拷贝到设备端
    hipMemcpy(d_a, h_a, arraySize * sizeof(float), hipMemcpyHostToDevice);

    // 设置线程块和网格大小
    int blockSize = 256;
    int gridSize = (arraySize + blockSize - 1) / blockSize;

    // 启动内核
    mul_Scalar_matrix<<<gridSize, blockSize>>>(d_a, value, d_c, arraySize);

    // 将结果从设备端拷贝回主机端
    hipMemcpy(h_c, d_c, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果（为了简化，此处不打印全部结果）
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_c[i]);
    }

    // 释放内存
    free(h_a);
    free(h_c);
    hipFree(d_a);
    hipFree(d_c);

    return 0;
}
