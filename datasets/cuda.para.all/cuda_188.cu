
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA核函数
__global__ void allMulInplaceKernel(double *arr, double alpha, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        arr[i] *= alpha;
    }
}

int main() {
    // 设置数据大小
    const int data_size = 100;
    const double alpha = 2.0;

    // 在设备上分配空间
    double *arr_device;
    hipMalloc((void**)&arr_device, data_size * sizeof(double));

    // 初始化数据
    double *arr_host = (double *)malloc(data_size * sizeof(double));
    for (int i = 0; i < data_size; ++i) {
        arr_host[i] = i + 1.0; // 为了演示目的，将输入数据初始化为 1.0, 2.0, 3.0, ...
    }

    // 将数据从主机复制到设备
    hipMemcpy(arr_device, arr_host, data_size * sizeof(double), hipMemcpyHostToDevice);

    // 定义启动配置
    dim3 blockDim(256); // 块大小为256个线程
    dim3 gridDim((data_size + blockDim.x - 1) / blockDim.x); // 确保足够的块数

    // 调用CUDA核函数
    allMulInplaceKernel<<<gridDim, blockDim>>>(arr_device, alpha, data_size);

    // 将结果从设备复制回主机
    hipMemcpy(arr_host, arr_device, data_size * sizeof(double), hipMemcpyDeviceToHost);

    // 打印结果
    printf("Result after CUDA kernel execution:\n");
    for (int i = 0; i < data_size; ++i) {
        printf("%.2f ", arr_host[i]);
    }
    printf("\n");

    // 释放内存
    free(arr_host);
    hipFree(arr_device);

    return 0;
}
 
