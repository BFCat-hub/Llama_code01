#include "hip/hip_runtime.h"
#include <>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NX 128
#define NY 64
#define B 512

// CUDA kernel
__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT) {
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    if (i < (NX + NY) * B) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        if (j < NX) {
            if (X)
                X[b * NX + j] += OUT[i];
        } else {
            if (Y)
                Y[b * NY + j - NX] += OUT[i];
        }
    }
}

int main() {
    // Allocate device memory
    float *d_X, *d_Y, *d_OUT;

    hipMalloc((void **)&d_X, B * NX * sizeof(float));
    hipMalloc((void **)&d_Y, B * NY * sizeof(float));
    hipMalloc((void **)&d_OUT, B * (NX + NY) * sizeof(float));

    // Set grid and block sizes
    dim3 blockSize(256);  // You may adjust the block size
    dim3 gridSize((B * (NX + NY) + blockSize.x - 1) / blockSize.x);

    // Launch the kernel
    deinter_kernel<<<gridSize, blockSize>>>(NX, d_X, NY, d_Y, B, d_OUT);

    // Synchronize to make sure the kernel finishes before proceeding
    hipDeviceSynchronize();

    // Check for errors during the kernel launch
    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaErr));
        return 1;
    }

    // Cleanup
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_OUT);

    return 0;
}
 
