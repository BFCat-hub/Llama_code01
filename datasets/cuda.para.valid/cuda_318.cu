
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean) {
    float scale = 1.f / (batch * spatial);
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters)
        return;

    int j, k;
    mean[i] = 0;

    for (j = 0; j < batch; ++j) {
        for (k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            mean[i] += x[index];
        }
    }

    mean[i] *= scale;
}

int main() {
    // Set the dimensions
    int batch = 64;      // Set the appropriate values
    int filters = 128;   // Set the appropriate values
    int spatial = 256;   // Set the appropriate values

    // Allocate device memory
    float *d_x, *d_mean;
    hipMalloc((void **)&d_x, batch * filters * spatial * sizeof(float));
    hipMalloc((void **)&d_mean, filters * sizeof(float));

    // Set grid and block sizes
    dim3 blockSize(256);  // You may adjust the block size
    dim3 gridSize((filters + blockSize.x - 1) / blockSize.x, 1);

    // Launch the kernel
    mean_kernel<<<gridSize, blockSize>>>(d_x, batch, filters, spatial, d_mean);

    // Synchronize to make sure the kernel finishes before proceeding
    hipDeviceSynchronize();

    // Check for errors during the kernel launch
    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaErr));
        return 1;
    }

    // Cleanup
    hipFree(d_x);
    hipFree(d_mean);

    return 0;
}
 
