
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA核函数
__global__ void vector_add(float *a, float *b, float *c) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    c[index] = a[index] + b[index];
}

int main() {
    // 设置数据大小
    const int data_size = 100;

    // 在设备上分配空间
    float *a_device, *b_device, *c_device;
    hipMalloc((void**)&a_device, data_size * sizeof(float));
    hipMalloc((void**)&b_device, data_size * sizeof(float));
    hipMalloc((void**)&c_device, data_size * sizeof(float));

    // 初始化输入数据
    float *a_host = (float *)malloc(data_size * sizeof(float));
    float *b_host = (float *)malloc(data_size * sizeof(float));
    for (int i = 0; i < data_size; ++i) {
        a_host[i] = i + 1.0; // 为了演示目的，将输入数据初始化为 1.0, 2.0, 3.0, ...
        b_host[i] = (i + 1.0) * 2.0; // 为了演示目的，将输入数据初始化为 2.0, 4.0, 6.0, ...
    }

    // 将输入数据从主机复制到设备
    hipMemcpy(a_device, a_host, data_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, data_size * sizeof(float), hipMemcpyHostToDevice);

    // 定义启动配置
    dim3 blockDim(256); // 块大小为256个线程
    dim3 gridDim((data_size + blockDim.x - 1) / blockDim.x); // 确保足够的块数

    // 调用CUDA核函数
    vector_add<<<gridDim, blockDim>>>(a_device, b_device, c_device);

    // 将结果从设备复制回主机
    float *c_host = (float *)malloc(data_size * sizeof(float));
    hipMemcpy(c_host, c_device, data_size * sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果
    printf("Result after CUDA kernel execution:\n");
    for (int i = 0; i < data_size; ++i) {
        printf("%.2f ", c_host[i]);
    }
    printf("\n");

    // 释放内存
    free(a_host);
    free(b_host);
    free(c_host);
    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);

    return 0;
}
 
