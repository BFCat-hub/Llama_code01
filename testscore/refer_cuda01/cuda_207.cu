
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA核函数
__global__ void dadd_matrix(double *a, double *b, double *c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    // 设置矩阵大小
    const int matrix_size = 100;

    // 在设备上分配空间
    double *a_device, *b_device, *c_device;
    hipMalloc((void**)&a_device, matrix_size * sizeof(double));
    hipMalloc((void**)&b_device, matrix_size * sizeof(double));
    hipMalloc((void**)&c_device, matrix_size * sizeof(double));

    // 初始化输入矩阵数据
    double *a_host = (double *)malloc(matrix_size * sizeof(double));
    double *b_host = (double *)malloc(matrix_size * sizeof(double));
    for (int i = 0; i < matrix_size; ++i) {
        a_host[i] = i + 1.0; // 为了演示目的，将输入数据初始化为 1.0, 2.0, 3.0, ...
        b_host[i] = (i + 1.0) * 2.0; // 为了演示目的，将输入数据初始化为 2.0, 4.0, 6.0, ...
    }

    // 将输入数据从主机复制到设备
    hipMemcpy(a_device, a_host, matrix_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, matrix_size * sizeof(double), hipMemcpyHostToDevice);

    // 定义启动配置
    dim3 blockDim(256); // 块大小为256个线程
    dim3 gridDim((matrix_size + blockDim.x - 1) / blockDim.x); // 确保足够的块数

    // 调用CUDA核函数
    dadd_matrix<<<gridDim, blockDim>>>(a_device, b_device, c_device, matrix_size);

    // 将结果从设备复制回主机
    double *c_result = (double *)malloc(matrix_size * sizeof(double));
    hipMemcpy(c_result, c_device, matrix_size * sizeof(double), hipMemcpyDeviceToHost);

    // 打印结果
    printf("Result after CUDA kernel execution:\n");
    for (int i = 0; i < matrix_size; ++i) {
        printf("%.2f ", c_result[i]);
    }
    printf("\n");

    // 释放内存
    free(a_host);
    free(b_host);
    free(c_result);
    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);

    return 0;
}
 
