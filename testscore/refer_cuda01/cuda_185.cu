
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA核函数
__global__ void setSuppressed(int *suppressed, int dims) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= dims) {
        return;
    }
    suppressed[tid] = 0;
}

int main() {
    // 设置数据大小
    const int data_size = 100;

    // 在设备上分配空间
    int *suppressed_device;
    hipMalloc((void**)&suppressed_device, data_size * sizeof(int));

    // 定义启动配置
    dim3 blockDim(256); // 块大小为256个线程
    dim3 gridDim((data_size + blockDim.x - 1) / blockDim.x); // 确保足够的块数

    // 调用CUDA核函数
    setSuppressed<<<gridDim, blockDim>>>(suppressed_device, data_size);

    // 将结果从设备复制回主机
    int *suppressed_host = (int *)malloc(data_size * sizeof(int));
    hipMemcpy(suppressed_host, suppressed_device, data_size * sizeof(int), hipMemcpyDeviceToHost);

    // 打印结果
    printf("Result after CUDA kernel execution:\n");
    for (int i = 0; i < data_size; ++i) {
        printf("%d ", suppressed_host[i]);
    }
    printf("\n");

    // 释放内存
    free(suppressed_host);
    hipFree(suppressed_device);

    return 0;
}
 
