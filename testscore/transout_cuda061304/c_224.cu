
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void vectorAdd(float* a, float* b, float* c, int size) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    
    int arraySize = 1000;

    
    float* h_a = (float*)malloc(arraySize * sizeof(float));
    float* h_b = (float*)malloc(arraySize * sizeof(float));
    float* h_c = (float*)malloc(arraySize * sizeof(float));

    
    for (int i = 0; i < arraySize; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(2 * i);
    }

    
    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc((void**)&d_a, arraySize * sizeof(float));
    hipMalloc((void**)&d_b, arraySize * sizeof(float));
    hipMalloc((void**)&d_c, arraySize * sizeof(float));

    
    hipMemcpy(d_a, h_a, arraySize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, arraySize * sizeof(float), hipMemcpyHostToDevice);

    
    int blockSize = 256;
    int gridSize = (arraySize + blockSize - 1) / blockSize;

    
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, arraySize);

    
    hipMemcpy(h_c, d_c, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_c[i]);
    }

    
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}