
#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void vecAdd(float* in1, float* in2, float* out, size_t size) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid < size) {
        out[gid] = in1[gid] + in2[gid];
    }
}

int main() {
    size_t size = 1000;

    float* h_in1 = (float*)malloc(size * sizeof(float));
    float* h_in2 = (float*)malloc(size * sizeof(float));
    float* h_out = (float*)malloc(size * sizeof(float));

    for (size_t i = 0; i < size; ++i) {
        h_in1[i] = static_cast<float>(i);
        h_in2[i] = static_cast<float>(2 * i);
    }

    float* d_in1;
    float* d_in2;
    float* d_out;
    hipMalloc((void**)&d_in1, size * sizeof(float));
    hipMalloc((void**)&d_in2, size * sizeof(float));
    hipMalloc((void**)&d_out, size * sizeof(float));

    hipMemcpy(d_in1, h_in1, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_in2, h_in2, size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize((size + blockSize.x - 1) / blockSize.x, 1);

    vecAdd<<<gridSize, blockSize>>>(d_in1, d_in2, d_out, size);

    hipMemcpy(h_out, d_out, size * sizeof(float), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < 10; ++i) {
        printf("%f ", h_out[i]);
    }

    free(h_in1);
    free(h_in2);
    free(h_out);
    hipFree(d_in1);
    hipFree(d_in2);
    hipFree(d_out);

    return 0;
}