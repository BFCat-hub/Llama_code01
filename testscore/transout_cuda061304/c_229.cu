
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void transferMBR3(double* xy_in, long long* a_out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        a_out[tid] = static_cast<long long>(xy_in[tid] * 10000000);
    }
}

int main() {
    int size = 1000;

    double* h_xy_in = (double*)malloc(size * sizeof(double));
    long long* h_a_out = (long long*)malloc(size * sizeof(long long));

    for (int i = 0; i < size; ++i) {
        h_xy_in[i] = static_cast<double>(i);
    }

    double* d_xy_in;
    long long* d_a_out;
    hipMalloc((void**)&d_xy_in, size * sizeof(double));
    hipMalloc((void**)&d_a_out, size * sizeof(long long));

    hipMemcpy(d_xy_in, h_xy_in, size * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    transferMBR3<<<gridSize, blockSize>>>(d_xy_in, d_a_out, size);

    hipMemcpy(h_a_out, d_a_out, size * sizeof(long long), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i) {
        printf("%ld ", h_a_out[i]);
    }

    free(h_xy_in);
    free(h_a_out);
    hipFree(d_xy_in);
    hipFree(d_a_out);

    return 0;
}