#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void saxpi(int n, float a, float* x, float* y) {
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

int main() {
    printf("SAXPI_CUDA sample\n");

    
    int n = 1000;
    float a = 0.1;

    
    float* h_x = (float*)malloc(n * sizeof(float));
    float* h_y = (float*)malloc(n * sizeof(float));

    
    for (int i = 0; i < n; ++i) {
        h_x[i] = static_cast<float>(i);
        h_y[i] = static_cast<float>(i * 2);
    }

    
    float* d_x;
    float* d_y;
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));

    
    hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);

    
    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, 1);

    
    saxpi<<<gridSize, blockSize>>>(n, a, d_x, d_y);

    
    hipMemcpy(h_y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < 10; ++i) {
        printf("y[%d]: %f\n", i, h_y[i]);
    }

    
    free(h_x);
    free(h_y);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}