
#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void vectorDiv(float* A, float* B, float* C, int numElements) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid < numElements) {
        C[gid] = A[gid] / B[gid];
    }
}

int main() {
    
    int numElements = 1000;

    
    float* h_A = (float*)malloc(numElements * sizeof(float));
    float* h_B = (float*)malloc(numElements * sizeof(float));
    float* h_C = (float*)malloc(numElements * sizeof(float));

    
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, numElements * sizeof(float));
    hipMalloc((void**)&d_B, numElements * sizeof(float));
    hipMalloc((void**)&d_C, numElements * sizeof(float));

    
    hipMemcpy(d_A, h_A, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, numElements * sizeof(float), hipMemcpyHostToDevice);

    
    int blockSize = 256;
    int gridSize = (numElements + blockSize - 1) / blockSize;

    
    vectorDiv<<<gridSize, blockSize>>>(d_A, d_B, d_C, numElements);

    
    hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_C[i]);
    }

    
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}