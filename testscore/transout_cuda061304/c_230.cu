
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void binarize_kernel(float* input, int n, float* binary) {
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    if (i < n) {
        binary[i] = (input[i] > 0) ? 1.0f : -1.0f;
    }
}

int main() {
    int n = 1000;

    float* h_input = (float*)malloc(n * sizeof(float));
    float* h_binary = (float*)malloc(n * sizeof(float));

    for (int i = 0; i < n; ++i) {
        h_input[i] = static_cast<float>(i);
    }

    float* d_input;
    float* d_binary;
    hipMalloc((void**)&d_input, n * sizeof(float));
    hipMalloc((void**)&d_binary, n * sizeof(float));

    hipMemcpy(d_input, h_input, n * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, 1);

    binarize_kernel<<<gridSize, blockSize>>>(d_input, n, d_binary);

    hipMemcpy(h_binary, d_binary, n * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i) {
        printf("h_binary[%d]: %f\n", i, h_binary[i]);
    }

    free(h_input);
    free(h_binary);
    hipFree(d_input);
    hipFree(d_binary);

    return 0;
}