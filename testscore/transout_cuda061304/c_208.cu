
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void test1(float* input, int dims) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= dims) {
        return;
    }

    if (input[tid * 4] != 0) {
        input[tid * 4] = 0;
    }
}

int main() {
    int dims = 1000;

    float* h_input = (float*)malloc(dims * 4 * sizeof(float));

    float* d_input;
    hipMalloc((void**)&d_input, dims * 4 * sizeof(float));

    hipMemcpy(d_input, h_input, dims * 4 * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (dims + blockSize - 1) / blockSize;

    test1<<<gridSize, blockSize>>>(d_input, dims);

    hipMemcpy(h_input, d_input, dims * 4 * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 3; ++i) {
        printf("h_input[%d]: %f\n", i, h_input[i]);
    }

    free(h_input);
    hipFree(d_input);

    return 0;
}