#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void clearArray(unsigned char* arr, const unsigned int length) {
    unsigned int offset = threadIdx.x + blockDim.x * blockIdx.x;
    while (offset < length) {
        arr[offset] = 0;
        offset += blockDim.x * gridDim.x;
    }
}

int main() {
    const unsigned int length = 1000;

    unsigned char* h_arr = (unsigned char*)malloc(length * sizeof(unsigned char));

    unsigned char* d_arr;
    hipMalloc((void**)&d_arr, length * sizeof(unsigned char));

    hipMemcpy(d_arr, h_arr, length * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize((length + blockSize.x - 1) / blockSize.x, 1);

    clearArray<<<gridSize, blockSize>>>(d_arr, length);

    hipMemcpy(h_arr, d_arr, length * sizeof(unsigned char), hipMemcpyDeviceToHost);

    for (unsigned int i = 0; i < 10; ++i) {
        printf("%u ", h_arr[i]);
    }

    free(h_arr);
    hipFree(d_arr);

    return 0;
}