
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void subtract_matrix(float* a, float* b, float* c, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        c[idx] = a[idx] - b[idx];
    }
}

int main() {
    
    int N = 1000;

    
    float* h_a = (float*)malloc(N * sizeof(float));
    float* h_b = (float*)malloc(N * sizeof(float));
    float* h_c = (float*)malloc(N * sizeof(float));

    
    for (int i = 0; i < N; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(2 * i);
    }

    
    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_c, N * sizeof(float));

    
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    
    subtract_matrix<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);

    
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_c[i]);
    }

    
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}