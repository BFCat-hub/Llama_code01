
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void VecAdd(float* A, float* B, float* C, int N) {
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;

    for (int i = 0; i < N; i += num_threads) {
        int index = i + thread_index;

        if (index < N) {
            C[index] = A[index] + B[index];
        }
    }
}

int main() {
    int array_size = 1000;

    float* h_array_A = (float*)malloc(array_size * sizeof(float));
    float* h_array_B = (float*)malloc(array_size * sizeof(float));
    float* h_array_C = (float*)malloc(array_size * sizeof(float));

    for (int i = 0; i < array_size; ++i) {
        h_array_A[i] = static_cast<float>(i);
        h_array_B[i] = static_cast<float>(2 * i);
    }

    float* d_array_A;
    float* d_array_B;
    float* d_array_C;
    hipMalloc((void**)&d_array_A, array_size * sizeof(float));
    hipMalloc((void**)&d_array_B, array_size * sizeof(float));
    hipMalloc((void**)&d_array_C, array_size * sizeof(float));

    hipMemcpy(d_array_A, h_array_A, array_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_array_B, h_array_B, array_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize((array_size + blockSize.x - 1) / blockSize.x, 1);

    VecAdd<<<gridSize, blockSize>>>(d_array_A, d_array_B, d_array_C, array_size);

    hipMemcpy(h_array_C, d_array_C, array_size * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_array_C[i]);
    }

    free(h_array_A);
    free(h_array_B);
    free(h_array_C);
    hipFree(d_array_A);
    hipFree(d_array_B);
    hipFree(d_array_C);

    return 0;
}