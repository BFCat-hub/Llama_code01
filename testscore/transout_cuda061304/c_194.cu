
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void histogram(int n, const int* color, int* bucket) {
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for (int i = 0; i < n; i += num_threads) {
        int c = color[i];
        atomicAdd(&bucket[c], 1);
    }
}

int main() {
    int n = 1000; 
    int num_buckets = 256;

    
    int* h_color = (int*)malloc(n * sizeof(int));
    int* h_bucket = (int*)malloc(num_buckets * sizeof(int));

    

    
    int* d_color, * d_bucket;
    hipMalloc((void**)&d_color, n * sizeof(int));
    hipMalloc((void**)&d_bucket, num_buckets * sizeof(int));

    
    hipMemcpy(d_color, h_color, n * sizeof(int), hipMemcpyHostToDevice);

    
    dim3 blockSize(256); 
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, 1);

    
    histogram<<<gridSize, blockSize>>>(n, d_color, d_bucket);

    
    hipMemcpy(h_bucket, d_bucket, num_buckets * sizeof(int), hipMemcpyDeviceToHost);

    

    
    free(h_color);
    free(h_bucket);
    hipFree(d_color);
    hipFree(d_bucket);

    return 0;
}