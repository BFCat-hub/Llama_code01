
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void gpu_add(float* c, float* a, float* b, int n) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (j < n) {
        c[j] = a[j] + b[j];
    }
}

int main() {
    
    int array_size = 1000;

    
    float* h_a = (float*)malloc(array_size * sizeof(float));
    float* h_b = (float*)malloc(array_size * sizeof(float));
    float* h_c = (float*)malloc(array_size * sizeof(float));

    
    for (int i = 0; i < array_size; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(2 * i);
    }

    
    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc((void**)&d_a, array_size * sizeof(float));
    hipMalloc((void**)&d_b, array_size * sizeof(float));
    hipMalloc((void**)&d_c, array_size * sizeof(float));

    
    hipMemcpy(d_a, h_a, array_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, array_size * sizeof(float), hipMemcpyHostToDevice);

    
    int blockSize = 256;
    int gridSize = (array_size + blockSize - 1) / blockSize;

    
    gpu_add<<<gridSize, blockSize>>>(d_c, d_a, d_b, array_size);

    
    hipMemcpy(h_c, d_c, array_size * sizeof(float), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_c[i]);
    }

    
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}