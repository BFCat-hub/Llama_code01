
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void subAvg(int* input, int count, int avg) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (index < count) {
        input[index] -= avg;
        index += stride;
    }
}

int main() {
    
    int count = 512;
    int avg = 0;

    
    int* h_input = (int*)malloc(count * sizeof(int));

    
    for (int i = 0; i < count; ++i) {
        h_input[i] = i;
    }

    
    int* d_input;
    hipMalloc((void**)&d_input, count * sizeof(int));

    
    hipMemcpy(d_input, h_input, count * sizeof(int), hipMemcpyHostToDevice);

    
    int blockSize = 256;
    int gridSize = (count + blockSize - 1) / blockSize;

    
    subAvg<<<gridSize, blockSize>>>(d_input, count, avg);

    
    hipMemcpy(h_input, d_input, count * sizeof(int), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < 10; ++i) {
        printf("%d ", h_input[i]);
    }

    
    free(h_input);
    hipFree(d_input);

    return 0;
}