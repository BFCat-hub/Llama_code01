#include "hip/hip_runtime.h"
#include <>
#include <stdio.h>


#define N 16

__global__ void vecAddGPU(float* input_a, float* input_b, float* output) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    output[idx] = input_a[idx] + input_b[idx];
}

int main() {
    
    float h_input_a[N];
    float h_input_b[N];
    float h_output[N];

    
    for (int i = 0; i < N; ++i) {
        h_input_a[i] = static_cast<float>(i);
        h_input_b[i] = static_cast<float>(2 * i);
    }

    
    float* d_input_a;
    float* d_input_b;
    float* d_output;
    hipMalloc((void**)&d_input_a, N * sizeof(float));
    hipMalloc((void**)&d_input_b, N * sizeof(float));
    hipMalloc((void**)&d_output, N * sizeof(float));

    
    hipMemcpy(d_input_a, h_input_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input_b, h_input_b, N * sizeof(float), hipMemcpyHostToDevice);

    
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, 1);

    
    vecAddGPU<<<gridSize, blockSize>>>(d_input_a, d_input_b, d_output);

    
    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < 8; ++i) {
        printf("h_output[%d]: %f\n", i, h_output[i]);
    }

    
    hipFree(d_input_a);
    hipFree(d_input_b);
    hipFree(d_output);

    return 0;
}

```