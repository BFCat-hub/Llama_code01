
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void gpu_record(float* p, float* seis_kt, int* Gxz, int ng) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < ng) {
        seis_kt[id] = p[Gxz[id]];
    }
}

int main() {
    
    int ng = 1000; 

    
    float* h_p = (float*)malloc(ng * sizeof(float));
    float* h_seis_kt = (float*)malloc(ng * sizeof(float));
    int* h_Gxz = (int*)malloc(ng * sizeof(int));

    
    float* d_p, * d_seis_kt;
    int* d_Gxz;
    hipMalloc((void**)&d_p, ng * sizeof(float));
    hipMalloc((void**)&d_seis_kt, ng * sizeof(float));
    hipMalloc((void**)&d_Gxz, ng * sizeof(int));

    
    hipMemcpy(d_p, h_p, ng * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Gxz, h_Gxz, ng * sizeof(int), hipMemcpyHostToDevice);

    
    dim3 blockSize(256); 
    dim3 gridSize((ng + blockSize.x - 1) / blockSize.x, 1);

    
    gpu_record<<<gridSize, blockSize>>>(d_p, d_seis_kt, d_Gxz, ng);

    
    hipMemcpy(h_seis_kt, d_seis_kt, ng * sizeof(float), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < 10; ++i) {
        printf("seis_kt[%d]: %f\n", i, h_seis_kt[i]);
    }

    
    free(h_p);
    free(h_seis_kt);
    free(h_Gxz);
    hipFree(d_p);
    hipFree(d_seis_kt);
    hipFree(d_Gxz);

    return 0;
}