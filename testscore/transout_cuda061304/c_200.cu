
#include <hip/hip_runtime.h>

#include <stdio.h>


__global__ void Copy_List_gpu(const int element_numbers, const float *origin_list, float *list) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < element_numbers; i += stride) {
        list[i] = origin_list[i];
    }
}

int main() {
    int element_numbers = 1000;

    
    float *origin_list, *list;

    
    hipSetDevice(0);

    
    float *d_origin_list, *d_list;
    hipMalloc((void **)&d_origin_list, element_numbers * sizeof(float));
    hipMalloc((void **)&d_list, element_numbers * sizeof(float));

    
    hipMemcpy(d_origin_list, origin_list, element_numbers * sizeof(float), hipMemcpyHostToDevice);

    
    int threadsPerBlock = 256;
    int blocksPerGrid = (element_numbers + threadsPerBlock - 1) / threadsPerBlock;

    
    Copy_List_gpu<<<blocksPerGrid, threadsPerBlock>>>(element_numbers, d_origin_list, d_list);

    
    hipDeviceSynchronize();

    
    hipMemcpy(list, d_list, element_numbers * sizeof(float), hipMemcpyDeviceToHost);

    
    hipFree(d_origin_list);
    hipFree(d_list);

    return 0;
}