
#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void vector_add(float* a, float* b, float* c) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    c[index] = a[index] + b[index];
}

int main() {
    
    int array_size = 1000;

    
    float* h_a = (float*)malloc(array_size * sizeof(float));
    float* h_b = (float*)malloc(array_size * sizeof(float));
    float* h_c = (float*)malloc(array_size * sizeof(float));

    
    for (int i = 0; i < array_size; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(2 * i);
    }

    
    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc((void**)&d_a, array_size * sizeof(float));
    hipMalloc((void**)&d_b, array_size * sizeof(float));
    hipMalloc((void**)&d_c, array_size * sizeof(float));

    
    hipMemcpy(d_a, h_a, array_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, array_size * sizeof(float), hipMemcpyHostToDevice);

    
    int block_size = 256;
    int grid_size = (array_size + block_size - 1) / block_size;

    
    vector_add<<<grid_size, block_size>>>(d_a, d_b, d_c);

    
    hipMemcpy(h_c, d_c, array_size * sizeof(float), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < 10; ++i) {
        printf("%f ", h_c[i]);
    }

    
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}